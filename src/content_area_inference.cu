#include <hip/hip_runtime.h>
#include "content_area_inference.cuh"

ContentAreaInference::ContentAreaInference()
{
    m_height_samples = 8;
    m_point_count = 2 * m_height_samples;

    hipMalloc(&m_dev_block, 2 * m_point_count * sizeof(uint));
    m_dev_points = m_dev_block;
    m_dev_scores = m_dev_block + m_point_count;
    
    m_hst_block = new uint[2 * m_point_count];
    m_hst_points = m_hst_block;
    m_hst_scores = m_hst_block + m_point_count;
}

ContentAreaInference::~ContentAreaInference()
{
    hipFree(m_dev_block);
    delete[] m_hst_block;
}

// Area ContentAreaInference::infer_area(uint8* image, const uint image_height, const uint image_width)
// {
//     ### Defined in "infer_area.cu" ###
// }

// void ContentAreaInference::draw_area(Area area, uint8* mask, const uint mask_height, const uint mask_width)
// {
//     ### Defined in "draw_area.cu" ###
// }
