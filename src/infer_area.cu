#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "content_area_inference.cuh"

#undef PROFILE

__device__  float normed_euclidean(uint8 r1, uint8 g1, uint8 b1, uint8 r2, uint8 g2, uint8 b2)
{
    #define EUCLID_NORM 441.67f // max possible value... sqrt(3 * 255 ^ 2)
    return sqrt((r1 - r2) * (r1 - r2) + (g1 - g2) * (g1 - g2) + (b1 - b2) * (b1 - b2)) / EUCLID_NORM;
}

template<int warp_count>
__global__ void find_points(uint8* g_image, uint* g_points, const uint image_width, const uint image_height, const uint height_gap, const uint point_count)
{
    __shared__ bool s_is_edge[warp_count];
    __shared__ uint s_indicies[warp_count];

    bool flip = blockIdx.x == 1;

    uint point_index = blockIdx.y;

    uint image_x = flip ? image_width - 1 - threadIdx.x : threadIdx.x;
    uint image_y = (point_index + 0.5) * height_gap;

    uint warp_index = threadIdx.x >> 5;
    uint lane_index = threadIdx.x & 31;
    uint neighbour_offset = flip ? -1 : 1;

    float edge_strength = normed_euclidean(
        g_image[image_x + image_y * image_width + 0 * image_width * image_height],
        g_image[image_x + image_y * image_width + 1 * image_width * image_height],
        g_image[image_x + image_y * image_width + 2 * image_width * image_height],
        g_image[image_x + neighbour_offset + image_y * image_width + 0 * image_width * image_height],
        g_image[image_x + neighbour_offset + image_y * image_width + 1 * image_width * image_height],
        g_image[image_x + neighbour_offset + image_y * image_width + 2 * image_width * image_height]
    );

    bool is_edge = edge_strength > 0.026;

    // ######################################
    // Finding first Edge above threshold...

    uint index = threadIdx.x;

    // Finding first edge in warp
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2)
    {
        bool other_is_edge = __shfl_down_sync(0xffffffff, is_edge, offset);
        uint other_index = __shfl_down_sync(0xffffffff, index, offset);

        if ((other_is_edge && other_index < index) || (other_is_edge && !is_edge))
        {
            is_edge = other_is_edge;
            index = other_index;
        }
    }

    // Writing first edge in warp to shared memory
    if (lane_index == 0)
    {    
        s_is_edge[warp_index] = is_edge;
        s_indicies[warp_index] = index;
    }

    // Syncing between warps
    __syncthreads();

    // Finding first edge in block
    if (warp_index == 0 && lane_index < warp_count)
    {
        is_edge = s_is_edge[lane_index];
        index = s_indicies[lane_index];

        #pragma unroll
        for (int offset = 16; offset > 0; offset /= 2)
        {
            bool other_is_edge = __shfl_down_sync(0xffffffff, is_edge, offset);
            uint other_index = __shfl_down_sync(0xffffffff, index, offset);

            if ((other_is_edge && other_index < index) || (other_is_edge && !is_edge))
            {
                is_edge = other_is_edge;
                index = other_index;
            }
        }

        // Saving result to global memory
        if (lane_index == 0)
        {
            int point_offset = flip ? point_count : 0;
            g_points[point_index + point_offset] = flip ? image_width - index - 1 : index;
        }
    }
}

__host__ __device__ bool calculate_circle(float ax, float ay, float bx, float by, float cx, float cy, float* x, float* y, float* r)
{
    float offset = bx * bx + by * by;

    float bc = 0.5f * (ax * ax + ay * ay - offset);
    float cd = 0.5f * (offset - cx * cx - cy * cy);

    float det = (ax - bx) * (by - cy) - (bx - cx) * (ay - by);

    bool valid = abs(det) > 1e-8; 

    if (valid)
    {
        float idet = 1.0f / det;

        *x = (bc * (by - cy) - cd * (ay - by)) * idet;
        *y = (cd * (ax - bx) - bc * (bx - cx)) * idet;
        *r = sqrt((bx - *x) * (bx - *x) + (by - *y) * (by - *y));
    }

    return valid;
}

__host__ __device__ uint triangle_size(const uint n)
{
    return n * (n - 1) / 2;
}

__device__ void square_indices(const int k, const int n, uint* i, uint* j)
{
    *i = n - 2 - int(sqrt(-8 * k + 4 * n * (n - 1) - 7) / 2.0 - 0.5);
    *j = k + *i + 1 -  n * (n - 1) / 2 + (n - *i) * ((n - *i) - 1) / 2;
}

__global__ void check_triples(const uint* g_points, uint* g_point_scores, const uint height_gap, const uint point_count, const uint image_height, const uint image_width)
{
    // HARDCODED SIZE!!!!!!!!!!!!!!!
    __shared__ uint s_points[32];
    __shared__ uint s_scores[32];

    if (threadIdx.x < point_count)
    {
        s_points[threadIdx.x] = g_points[threadIdx.x];
    }

    const uint warp_count = (blockDim.x >> 5) + 1; /// +1 ????????????????
    const uint warp_index = threadIdx.x >> 5;
    const uint lane_index = threadIdx.x & 31;

    uint a_index, b_index, c_index;
    a_index = blockIdx.x;
    square_indices(threadIdx.x, point_count, &b_index, &c_index);

    float ax = s_points[a_index];
    float bx = s_points[b_index];
    float cx = s_points[c_index];

    float ay = height_gap * (0.5  + (a_index % (point_count / 2))); 
    float by = height_gap * (0.5  + (b_index % (point_count / 2)));
    float cy = height_gap * (0.5  + (c_index % (point_count / 2)));

    float x, y, r;
    bool valid = calculate_circle(ax, ay, bx, by, cx, cy, &x, &y, &r);

    // Filter out bad circles
    uint score = valid; 
    score &= abs(x - 0.5 * image_width) < 0.1 * image_width;
    score &= abs(y - 0.5 * image_height) < 0.1 * image_height;
    score &= r > (0.3 * image_width);
    score &= r < (0.6 * image_width);

    // Warp reduction
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2)
    {
        score += __shfl_down_sync(0xffffffff, score, offset);
    }

    if (lane_index == 0)
    { 
        s_scores[warp_index] = score;
    }

    // Syncing between warps
    __syncthreads();

    // Block reduction
    if (warp_index == 0 && lane_index < warp_count)
    {
        score = s_scores[lane_index];
        
        #pragma unroll
        for (int offset = warp_count / 2; offset > 0; offset /= 2)
        {
            score += __shfl_down_sync(0xffffffff, score, offset);
        }

        // Outputting result
        if (lane_index == 0)
        {
            g_point_scores[a_index] = score;
        }
    }
}

float distance_score(const uint point_count, const uint i, const uint j)
{
    float x_diff = ((i > point_count) != (j > point_count)); 
    float y_diff = abs(float(i) - j) / point_count;

    return sqrt((x_diff * x_diff + y_diff * y_diff) / 2);
}

void select_final_triple(const uint point_count, const uint* scores, int* indices)
{
    float best_value = 0.0f;
    float best_score = 0.0f;

    for (int i = 0; i < point_count; i++)
    {
        float score = scores[i];


        if (score > best_score)
        {
            best_score = score;
        }
    }

    for (int i = 0; i < point_count; i++)
    {
        float score_i = scores[i];

        for (int j = i+1; j < point_count; j++)
        {
            float score_j = scores[j];

            for (int k = j+1; k < point_count; k++)
            {
                float score_k = scores[k];

                float dist_value = distance_score(point_count, i, j) + distance_score(point_count, i, k) + distance_score(point_count, j, k);
                float score_value = score_i * score_j * score_k / (3 * best_score * best_score * best_score);

                float value = dist_value * score_value;

                if (value > best_value)
                {
                    best_value = value;
                    
                    indices[0] = i;
                    indices[1] = j;
                    indices[2] = k;
                }
            }
        }
    }
}

#define warp_size 32
#define warp_count 16

Area ContentAreaInference::infer_area(uint8* image, const uint image_height, const uint image_width)
{
    #ifdef PROFILE
    hipEvent_t a, b, c, d, e;
    hipEventCreate(&a);
    hipEventCreate(&b);
    hipEventCreate(&c);
    hipEventCreate(&d);
    hipEventCreate(&e);
    #endif

    // #########################################################
    // Some useful values...

    uint height_gap = image_height / m_height_samples;

    // #########################################################
    // Finding candididate points...
    // A thread block for each point
    #ifdef PROFILE
    hipEventRecord(a);
    #endif  

    dim3 find_points_grid(2, m_height_samples);
    dim3 find_points_block(warp_size * warp_count, 1);
    find_points<warp_count><<<find_points_grid, find_points_block>>>(image, m_dev_points, image_width, image_height, height_gap, m_height_samples);
    
    // #########################################################
    // Evaluating candidate points...
    // A thread block for each point (left and right)
    // A thread per combination of the other two points in each triple.
    #ifdef PROFILE
    hipEventRecord(b);
    #endif  

    dim3 check_triples_grid(m_point_count);
    dim3 check_triples_block(triangle_size(m_point_count));
    check_triples<<<check_triples_grid, check_triples_block>>>(m_dev_points, m_dev_scores, height_gap, m_point_count, image_height, image_width);

    // #########################################################
    // Reading back results and freeing cuda memory...
    #ifdef PROFILE
    hipEventRecord(c);
    #endif  

    hipMemcpy(m_hst_block, m_dev_block, 2 * m_point_count * sizeof(uint), hipMemcpyDeviceToHost);

    // #########################################################
    // Choosing the final points and calculating circle...
    #ifdef PROFILE
    hipEventRecord(d);
    #endif  

    int indices[3];
    select_final_triple(m_point_count, m_hst_scores, indices);

    float ax = m_hst_points[indices[0]];
    float bx = m_hst_points[indices[1]];
    float cx = m_hst_points[indices[2]];

    float ay = int(((indices[0] % m_height_samples) + 0.5) * height_gap);
    float by = int(((indices[1] % m_height_samples) + 0.5) * height_gap);
    float cy = int(((indices[2] % m_height_samples) + 0.5) * height_gap);

    float x, y, r;
    calculate_circle(ax, ay, bx, by, cx, cy, &x, &y, &r);

    // #########################################################
    // Constructing final area to return...
    #ifdef PROFILE
    hipEventRecord(e);

    hipEventSynchronize(a);
    hipEventSynchronize(b);
    hipEventSynchronize(c);
    hipEventSynchronize(d);
    hipEventSynchronize(e);

    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, a, e);
    ADD_SAMPLE("infer area", milliseconds);
    hipEventElapsedTime(&milliseconds, a, b);
    ADD_SAMPLE("infer area: find points", milliseconds);
    hipEventElapsedTime(&milliseconds, b, c);
    ADD_SAMPLE("infer area: check triples", milliseconds);
    hipEventElapsedTime(&milliseconds, c, d);
    ADD_SAMPLE("infer area: read back points", milliseconds);
    hipEventElapsedTime(&milliseconds, d, e);
    ADD_SAMPLE("infer area: choose final points", milliseconds);
    #endif  

    Area area;
    area.type = Area::Circle;
    area.circle.y = y;
    area.circle.x = x;
    area.circle.r = r;

    return area;
}
