#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "content_area.h"

#include <iostream>

__device__ uint8 grayscale(uint8 r, uint8 g, uint8 b)
{
    return 0.2989 * r + 0.5870 * g + 0.1140 * b;
}

template<int warp_count>
__global__ void find_points(uint8* g_image, uint* g_points, const uint image_width, const uint image_height, const uint height_gap, const uint point_count)
{
    __shared__ bool s_is_edge[warp_count];
    __shared__ uint s_indicies[warp_count];

    bool flip = blockIdx.x == 1;

    uint point_index = blockIdx.y;

    uint image_x = flip ? image_width - 1 - threadIdx.x : threadIdx.x;
    uint image_y = (point_index + 0.5) * height_gap;

    uint warp_index = threadIdx.x >> 5;
    uint lane_index = threadIdx.x & 31;

    int home = grayscale(
        g_image[image_x + image_y * image_width + 0 * image_width * image_height],
        g_image[image_x + image_y * image_width + 1 * image_width * image_height],
        g_image[image_x + image_y * image_width + 2 * image_width * image_height]
    );

    uint neighbour_offset = flip ? -1 : 1;

    int neighbour = grayscale(
        g_image[image_x + neighbour_offset + image_y * image_width + 0 * image_width * image_height],
        g_image[image_x + neighbour_offset + image_y * image_width + 1 * image_width * image_height],
        g_image[image_x + neighbour_offset + image_y * image_width + 2 * image_width * image_height]
    );

    bool is_edge = abs(home - neighbour) > 10;
    uint index = threadIdx.x;

    // Finding warp max
    for (int offset = 16; offset > 0; offset /= 2)
    {
        bool other_is_edge = __shfl_down_sync(0xffffffff, is_edge, offset);
        uint other_index = __shfl_down_sync(0xffffffff, index, offset);

        if ((other_is_edge && other_index < index) || (other_is_edge && !is_edge))
        {
            is_edge = other_is_edge;
            index = other_index;
        }
    }

    // Writing warp max to shared memory
    if (lane_index == 0)
    {    
        s_is_edge[warp_index] = is_edge;
        s_indicies[warp_index] = index;
    }

    // Syncing between warps
    __syncthreads();

    // Finding block max
    if (warp_index == 0 && lane_index < warp_count)
    {
        is_edge = s_is_edge[lane_index];
        index = s_indicies[lane_index];

        for (int offset = 16; offset > 0; offset /= 2)
        {
            bool other_is_edge = __shfl_down_sync(0xffffffff, is_edge, offset);
            uint other_index = __shfl_down_sync(0xffffffff, index, offset);

            if ((other_is_edge && other_index < index) || (other_is_edge && !is_edge))
            {
                is_edge = other_is_edge;
                index = other_index;
            }
        }

        // Outputting result
        if (lane_index == 0)
        {
            int point_offset = flip ? point_count : 0;
            g_points[point_index + point_offset] = flip ? image_width - index - 1 : index;
        }
    }
}

__host__ __device__ bool calculate_circle(float ax, float ay, float bx, float by, float cx, float cy, float* x, float* y, float* r)
{
    float offset = bx * bx + by * by;

    float bc = 0.5f * (ax * ax + ay * ay - offset);
    float cd = 0.5f * (offset - cx * cx - cy * cy);

    float det = (ax - bx) * (by - cy) - (bx - cx) * (ay - by);

    bool valid = abs(det) > 1e-8; 

    if (valid)
    {
        float idet = 1.0f / det;

        *x = (bc * (by - cy) - cd * (ay - by)) * idet;
        *y = (cd * (ax - bx) - bc * (bx - cx)) * idet;
        *r = sqrt((bx - *x) * (bx - *x) + (by - *y) * (by - *y));
    }

    return valid;
}

__host__ __device__ uint triangle_size(const uint n)
{
    return n * (n - 1) / 2;
}

__device__ void square_indices(const uint k, const uint n, uint* i, uint* j)
{
    *i = n - 2 - int(sqrt(-8 * k + 4 * n * (n - 1) - 7) / 2.0 - 0.5);
    *j = k + *i + 1 -  n * (n - 1) / 2 + (n - *i) * ((n - *i) - 1) / 2;
}

__global__ void check_triples(const uint* g_points, uint* g_point_scores, const uint height_gap, const uint point_count, const uint image_height, const uint image_width)
{
    // HARDCODED SIZE!!!!!!!!!!!!!!!
    __shared__ uint s_points[32];
    __shared__ uint s_scores[32];

    if (threadIdx.x < point_count)
    {
        s_points[threadIdx.x] = g_points[threadIdx.x];
    }

    const uint warp_count = (blockDim.x >> 5) + 1; /// +1 ????????????????
    const uint warp_index = threadIdx.x >> 5;
    const uint lane_index = threadIdx.x & 31;

    uint a_index, b_index, c_index;
    a_index = blockIdx.x;
    square_indices(threadIdx.x, point_count, &b_index, &c_index);

    float ax = s_points[a_index];
    float bx = s_points[b_index];
    float cx = s_points[c_index];

    float ay = height_gap * (0.5  + (a_index % (point_count / 2))); 
    float by = height_gap * (0.5  + (b_index % (point_count / 2)));
    float cy = height_gap * (0.5  + (c_index % (point_count / 2)));

    float x, y, r;
    bool valid = calculate_circle(ax, ay, bx, by, cx, cy, &x, &y, &r);

    // Filter out bad circles
    uint score = valid; 
    score &= abs(x - 0.5 * image_width) < 0.1 * image_width;
    score &= abs(y - 0.5 * image_height) < 0.1 * image_height;
    score &= r > (0.3 * image_width);
    score &= r < (0.6 * image_width);

    // Warp reduction
    for (int offset = 16; offset > 0; offset /= 2)
    {
        score += __shfl_down_sync(0xffffffff, score, offset);
    }

    if (lane_index == 0)
    { 
        s_scores[warp_index] = score;
    }

    // Syncing between warps
    __syncthreads();

    // Block reduction
    if (warp_index == 0 && lane_index < warp_count)
    {
        score = s_scores[lane_index];

        for (int offset = warp_count / 2; offset > 0; offset /= 2)
        {
            score += __shfl_down_sync(0xffffffff, score, offset);
        }

        // Outputting result
        if (lane_index == 0)
        {
            g_point_scores[a_index] = score;
        }
    }
}

float distance_score(const uint point_count, const uint i, const uint j)
{
    float x_diff = ((i > point_count) != (j > point_count)); 
    float y_diff = abs(float(i) - j) / point_count;

    return sqrt((x_diff * x_diff + y_diff * y_diff) / 2);
}

void select_final_triple(const uint point_count, const uint* scores, int* indices)
{
    float best_value = 0.0f;
    float best_score = 0.0f;

    for (int i = 0; i < point_count; i++)
    {
        float score = scores[i];


        if (score > best_score)
        {
            best_score = score;
        }
    }

    for (int i = 0; i < point_count; i++)
    {
        float score_i = scores[i];

        for (int j = i+1; j < point_count; j++)
        {
            float score_j = scores[j];

            for (int k = j+1; k < point_count; k++)
            {
                float score_k = scores[k];

                float dist_value = distance_score(point_count, i, j) + distance_score(point_count, i, k) + distance_score(point_count, j, k);
                float score_value = score_i * score_j * score_k / (3 * best_score * best_score * best_score);

                float value = dist_value * score_value;

                if (value > best_value)
                {
                    best_value = value;
                    
                    indices[0] = i;
                    indices[1] = j;
                    indices[2] = k;
                }
            }
        }
    }
}

#define warp_size 32
#define warp_count 16

Area infer_area_cuda(uint8* image, const uint image_height, const uint image_width, const uint height_samples)
{
    // #########################################################
    // Some useful values...

    uint point_count = 2 * height_samples;
    uint height_gap = image_height / height_samples;
    
    // #########################################################
    // Allocating cuda memory...

    uint* dev_points;
    hipMalloc(&dev_points, point_count * sizeof(uint));

    uint* dev_point_scores;
    hipMalloc(&dev_point_scores, point_count * sizeof(uint));

    // #########################################################
    // Finding candididate points...
    // A thread block for each point

    dim3 find_points_grid(2, height_samples);
    dim3 find_points_block(warp_size * warp_count, 1);
    find_points<warp_count><<<find_points_grid, find_points_block>>>(image, dev_points, image_width, image_height, height_gap, height_samples);
    
    // #########################################################
    // Evaluating candidate points...
    // A thread block for each point (left and right)
    // A thread per combination of the other two points in each triple.

    dim3 check_triples_grid(point_count);
    dim3 check_triples_block(triangle_size(point_count));
    check_triples<<<check_triples_grid, check_triples_block>>>(dev_points, dev_point_scores, height_gap, point_count, image_height, image_width);

    // #########################################################
    // Reading back results and freeing cuda memory...

    uint* hst_points = new uint[point_count];
    hipMemcpy(hst_points, dev_points, point_count * sizeof(uint), hipMemcpyDeviceToHost);

    uint* hst_point_scores = new uint[point_count];
    hipMemcpy(hst_point_scores, dev_point_scores, point_count * sizeof(uint), hipMemcpyDeviceToHost);

    // #########################################################
    // Choosing the final points and calculating circle...

    int indices[3];
    select_final_triple(point_count, hst_point_scores, indices);

    float ax = hst_points[indices[0]];
    float bx = hst_points[indices[1]];
    float cx = hst_points[indices[2]];

    float ay = int(((indices[0] % height_samples) + 0.5) * height_gap);
    float by = int(((indices[1] % height_samples) + 0.5) * height_gap);
    float cy = int(((indices[2] % height_samples) + 0.5) * height_gap);

    float x, y, r;
    calculate_circle(ax, ay, bx, by, cx, cy, &x, &y, &r);

    // #########################################################
    // Freeing cuda and cpu memory... 

    hipFree(dev_point_scores);  
    hipFree(dev_points);

    delete hst_point_scores;
    delete hst_points;

    // #########################################################
    // Constructing final area to return...

    Area area;
    area.type = Area::Circle;
    area.circle.y = y;
    area.circle.x = x;
    area.circle.r = r;

    return area;
}





























// #define BLOCK_SIZE 32
// #define GRID_SIZE(d) ((d / BLOCK_SIZE) + 1)

// __global__ void gray_kernel(uint8* g_image, uint8* g_mask, const uint image_height, const uint image_width)
// {    
//     uint image_x = threadIdx.x + blockIdx.x * blockDim.x;
//     uint image_y = threadIdx.y + blockIdx.y * blockDim.y;

//     if (image_x > image_width || image_y > image_height)
//     {
//         return;
//     }

//     uint8 gray = grayscale(
//         g_image[image_x + image_y * image_width + 0 * image_width * image_height],
//         g_image[image_x + image_y * image_width + 1 * image_width * image_height],
//         g_image[image_x + image_y * image_width + 2 * image_width * image_height]
//     );

//     g_mask[image_x + image_y * image_width] = gray;
// }

// void gray(uint8* image, uint8* mask, const uint mask_height, const uint mask_width)
// {
//     dim3 grid(GRID_SIZE(mask_width), GRID_SIZE(mask_height));
//     dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    
//     std::cout << grid.x << " " << grid.y << std::endl;

//     gray_kernel<<<grid, block>>>(image, mask, mask_height, mask_width);
// }

