#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "content_area_inference.cuh"

__device__ uint8 grayscale(uint8 r, uint8 g, uint8 b)
{
    return 0.2989 * r + 0.5870 * g + 0.1140 * b;
}

template<int warp_count>
__global__ void find_points(uint8* g_image, uint* g_points, const uint image_width, const uint image_height, const uint height_gap, const uint point_count)
{
    __shared__ bool s_is_edge[warp_count];
    __shared__ uint s_indicies[warp_count];

    bool flip = blockIdx.x == 1;

    uint point_index = blockIdx.y;

    uint image_x = flip ? image_width - 1 - threadIdx.x : threadIdx.x;
    uint image_y = (point_index + 0.5) * height_gap;

    uint warp_index = threadIdx.x >> 5;
    uint lane_index = threadIdx.x & 31;

    int home = grayscale(
        g_image[image_x + image_y * image_width + 0 * image_width * image_height],
        g_image[image_x + image_y * image_width + 1 * image_width * image_height],
        g_image[image_x + image_y * image_width + 2 * image_width * image_height]
    );

    uint neighbour_offset = flip ? -1 : 1;

    int neighbour = grayscale(
        g_image[image_x + neighbour_offset + image_y * image_width + 0 * image_width * image_height],
        g_image[image_x + neighbour_offset + image_y * image_width + 1 * image_width * image_height],
        g_image[image_x + neighbour_offset + image_y * image_width + 2 * image_width * image_height]
    );

    bool is_edge = abs(home - neighbour) > 6;
    uint index = threadIdx.x;

    // Finding warp max
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2)
    {
        bool other_is_edge = __shfl_down_sync(0xffffffff, is_edge, offset);
        uint other_index = __shfl_down_sync(0xffffffff, index, offset);

        if ((other_is_edge && other_index < index) || (other_is_edge && !is_edge))
        {
            is_edge = other_is_edge;
            index = other_index;
        }
    }

    // Writing warp max to shared memory
    if (lane_index == 0)
    {    
        s_is_edge[warp_index] = is_edge;
        s_indicies[warp_index] = index;
    }

    // Syncing between warps
    __syncthreads();

    // Finding block max
    if (warp_index == 0 && lane_index < warp_count)
    {
        is_edge = s_is_edge[lane_index];
        index = s_indicies[lane_index];

        #pragma unroll
        for (int offset = 16; offset > 0; offset /= 2)
        {
            bool other_is_edge = __shfl_down_sync(0xffffffff, is_edge, offset);
            uint other_index = __shfl_down_sync(0xffffffff, index, offset);

            if ((other_is_edge && other_index < index) || (other_is_edge && !is_edge))
            {
                is_edge = other_is_edge;
                index = other_index;
            }
        }

        // Outputting result
        if (lane_index == 0)
        {
            int point_offset = flip ? point_count : 0;
            g_points[point_index + point_offset] = flip ? image_width - index - 1 : index;
        }
    }
}

__host__ __device__ bool calculate_circle(float ax, float ay, float bx, float by, float cx, float cy, float* x, float* y, float* r)
{
    float offset = bx * bx + by * by;

    float bc = 0.5f * (ax * ax + ay * ay - offset);
    float cd = 0.5f * (offset - cx * cx - cy * cy);

    float det = (ax - bx) * (by - cy) - (bx - cx) * (ay - by);

    bool valid = abs(det) > 1e-8; 

    if (valid)
    {
        float idet = 1.0f / det;

        *x = (bc * (by - cy) - cd * (ay - by)) * idet;
        *y = (cd * (ax - bx) - bc * (bx - cx)) * idet;
        *r = sqrt((bx - *x) * (bx - *x) + (by - *y) * (by - *y));
    }

    return valid;
}

__host__ __device__ uint triangle_size(const uint n)
{
    return n * (n - 1) / 2;
}

__device__ void square_indices(const uint k, const uint n, uint* i, uint* j)
{
    *i = n - 2 - int(sqrt(-8 * k + 4 * n * (n - 1) - 7) / 2.0 - 0.5);
    *j = k + *i + 1 -  n * (n - 1) / 2 + (n - *i) * ((n - *i) - 1) / 2;
}

__global__ void check_triples(const uint* g_points, uint* g_point_scores, const uint height_gap, const uint point_count, const uint image_height, const uint image_width)
{
    // HARDCODED SIZE!!!!!!!!!!!!!!!
    __shared__ uint s_points[32];
    __shared__ uint s_scores[32];

    if (threadIdx.x < point_count)
    {
        s_points[threadIdx.x] = g_points[threadIdx.x];
    }

    const uint warp_count = (blockDim.x >> 5) + 1; /// +1 ????????????????
    const uint warp_index = threadIdx.x >> 5;
    const uint lane_index = threadIdx.x & 31;

    uint a_index, b_index, c_index;
    a_index = blockIdx.x;
    square_indices(threadIdx.x, point_count, &b_index, &c_index);

    float ax = s_points[a_index];
    float bx = s_points[b_index];
    float cx = s_points[c_index];

    float ay = height_gap * (0.5  + (a_index % (point_count / 2))); 
    float by = height_gap * (0.5  + (b_index % (point_count / 2)));
    float cy = height_gap * (0.5  + (c_index % (point_count / 2)));

    float x, y, r;
    bool valid = calculate_circle(ax, ay, bx, by, cx, cy, &x, &y, &r);

    // Filter out bad circles
    uint score = valid; 
    score &= abs(x - 0.5 * image_width) < 0.1 * image_width;
    score &= abs(y - 0.5 * image_height) < 0.1 * image_height;
    score &= r > (0.3 * image_width);
    score &= r < (0.6 * image_width);

    // Warp reduction
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2)
    {
        score += __shfl_down_sync(0xffffffff, score, offset);
    }

    if (lane_index == 0)
    { 
        s_scores[warp_index] = score;
    }

    // Syncing between warps
    __syncthreads();

    // Block reduction
    if (warp_index == 0 && lane_index < warp_count)
    {
        score = s_scores[lane_index];
        
        #pragma unroll
        for (int offset = warp_count / 2; offset > 0; offset /= 2)
        {
            score += __shfl_down_sync(0xffffffff, score, offset);
        }

        // Outputting result
        if (lane_index == 0)
        {
            g_point_scores[a_index] = score;
        }
    }
}

float distance_score(const uint point_count, const uint i, const uint j)
{
    float x_diff = ((i > point_count) != (j > point_count)); 
    float y_diff = abs(float(i) - j) / point_count;

    return sqrt((x_diff * x_diff + y_diff * y_diff) / 2);
}

void select_final_triple(const uint point_count, const uint* scores, int* indices)
{
    float best_value = 0.0f;
    float best_score = 0.0f;

    for (int i = 0; i < point_count; i++)
    {
        float score = scores[i];


        if (score > best_score)
        {
            best_score = score;
        }
    }

    for (int i = 0; i < point_count; i++)
    {
        float score_i = scores[i];

        for (int j = i+1; j < point_count; j++)
        {
            float score_j = scores[j];

            for (int k = j+1; k < point_count; k++)
            {
                float score_k = scores[k];

                float dist_value = distance_score(point_count, i, j) + distance_score(point_count, i, k) + distance_score(point_count, j, k);
                float score_value = score_i * score_j * score_k / (3 * best_score * best_score * best_score);

                float value = dist_value * score_value;

                if (value > best_value)
                {
                    best_value = value;
                    
                    indices[0] = i;
                    indices[1] = j;
                    indices[2] = k;
                }
            }
        }
    }
}

#define warp_size 32
#define warp_count 16

// #define PROFILE

#ifdef PROFILE
#include <torch/extension.h>
#endif

Area ContentAreaInference::infer_area(uint8* image, const uint image_height, const uint image_width)
{
    #ifdef PROFILE
    hipEvent_t a, b, c, d, e;
    hipEventCreate(&a);
    hipEventCreate(&b);
    hipEventCreate(&c);
    hipEventCreate(&d);
    hipEventCreate(&e);
    #endif

    // #########################################################
    // Some useful values...

    uint height_gap = image_height / m_height_samples;

    // #########################################################
    // Finding candididate points...
    // A thread block for each point
    #ifdef PROFILE
    hipEventRecord(a);
    #endif  

    dim3 find_points_grid(2, m_height_samples);
    dim3 find_points_block(warp_size * warp_count, 1);
    find_points<warp_count><<<find_points_grid, find_points_block>>>(image, m_dev_points, image_width, image_height, height_gap, m_height_samples);
    
    // #########################################################
    // Evaluating candidate points...
    // A thread block for each point (left and right)
    // A thread per combination of the other two points in each triple.
    #ifdef PROFILE
    hipEventRecord(b);
    #endif  

    dim3 check_triples_grid(m_point_count);
    dim3 check_triples_block(triangle_size(m_point_count));
    check_triples<<<check_triples_grid, check_triples_block>>>(m_dev_points, m_dev_scores, height_gap, m_point_count, image_height, image_width);

    // #########################################################
    // Reading back results and freeing cuda memory...
    #ifdef PROFILE
    hipEventRecord(c);
    #endif  

    hipMemcpy(m_hst_block, m_dev_block, 2 * m_point_count * sizeof(uint), hipMemcpyDeviceToHost);

    // #########################################################
    // Choosing the final points and calculating circle...
    #ifdef PROFILE
    hipEventRecord(d);
    #endif  

    int indices[3];
    select_final_triple(m_point_count, m_hst_scores, indices);

    float ax = m_hst_points[indices[0]];
    float bx = m_hst_points[indices[1]];
    float cx = m_hst_points[indices[2]];

    float ay = int(((indices[0] % m_height_samples) + 0.5) * height_gap);
    float by = int(((indices[1] % m_height_samples) + 0.5) * height_gap);
    float cy = int(((indices[2] % m_height_samples) + 0.5) * height_gap);

    float x, y, r;
    calculate_circle(ax, ay, bx, by, cx, cy, &x, &y, &r);

    // #########################################################
    // Constructing final area to return...
    #ifdef PROFILE
    hipEventRecord(e);

    hipEventSynchronize(a);
    hipEventSynchronize(b);
    hipEventSynchronize(c);
    hipEventSynchronize(d);
    hipEventSynchronize(e);

    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, a, g);
    py::print("TOTAL:", milliseconds);
    hipEventElapsedTime(&milliseconds, a, b);
    py::print("find points:", milliseconds);
    hipEventElapsedTime(&milliseconds, b, c);
    py::print("check triples:", milliseconds);
    hipEventElapsedTime(&milliseconds, c, d);
    py::print("read back points:", milliseconds);
    hipEventElapsedTime(&milliseconds, d, e);
    py::print("choose final points:", milliseconds);
    #endif  

    Area area;
    area.type = Area::Circle;
    area.circle.y = y;
    area.circle.x = x;
    area.circle.r = r;

    return area;
}
