#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../common.hpp"

namespace cuda
{
    template<int warp_count>
    __global__ void find_best_edge(const float* g_score_strips, float* g_edge_x, float* g_edge_y, float* g_edge_scores, const int image_width, const int image_height, const int strip_count, const int half_patch_size)
    {
        __shared__ float s_cross_warp_operation_buffer[warp_count];
        __shared__ float s_cross_warp_operation_buffer_2[warp_count];

        int warp_index = threadIdx.x >> 5;
        int lane_index = threadIdx.x & 31;

        bool flip = blockIdx.x == 1;

        // ============================================================
        // Load strip into shared memory...

        int image_x = flip ? image_width - 1 - threadIdx.x : threadIdx.x;

        int strip_index = blockIdx.y;
        int strip_height = 1 + (image_height - 2) / (1.0f + exp(-(strip_index - strip_count / 2.0f + 0.5f)/(strip_count / 8.0f)));
        
        float point_score = g_score_strips[image_x + strip_index * image_width];
    
        int best_edge_x = image_x;
        float best_edge_score = point_score;
        
        // warp reduction....
        #pragma unroll
        for (int offset = 32 >> 1; offset > 0; offset >>= 1)
        {
            int other_edge_x = __shfl_down_sync(0xffffffff, best_edge_x, offset);
            float other_edge_score = __shfl_down_sync(0xffffffff, best_edge_score, offset);

            if (other_edge_score > best_edge_score)
            {
                best_edge_x = other_edge_x;
                best_edge_score = other_edge_score;
            }
        }

        if (lane_index == 0)
        {
            s_cross_warp_operation_buffer[warp_index] = best_edge_x;
            s_cross_warp_operation_buffer_2[warp_index] = best_edge_score;
        }

        __syncthreads();

        // block reduction....
        if (warp_index == 0 && lane_index < warp_count)
        {
            best_edge_x = s_cross_warp_operation_buffer[lane_index];
            best_edge_score = s_cross_warp_operation_buffer_2[lane_index];

            #pragma unroll
            for (int offset = warp_count >> 1 ; offset > 0; offset >>= 1)
            {
                int other_edge_x = __shfl_down_sync(0xffffffff, best_edge_x, offset);
                float other_edge_score = __shfl_down_sync(0xffffffff, best_edge_score, offset);

                if (other_edge_score > best_edge_score)
                {
                    best_edge_x = other_edge_x;
                    best_edge_score = other_edge_score;
                }
            }

            if (lane_index == 0)
            {
                int point_index = flip ? strip_index : strip_index + strip_count;
                g_edge_x[point_index] = best_edge_x + half_patch_size;
                g_edge_y[point_index] = strip_height;
                g_edge_scores[point_index] = best_edge_score;
            }
        }
    }

    #define warp_size 32
    #define warp_count 8

    void find_points_from_strip_scores(const float* strips, const int image_height, const int image_width, const int strip_count, const int model_patch_size, float* points_x, float* points_y, float* point_score)
    {
        int half_patch_size = (model_patch_size - 1) / 2;

        dim3 find_points_grid(2, strip_count);
        dim3 find_points_block(warp_size * warp_count);
        find_best_edge<warp_count><<<find_points_grid, find_points_block>>>(strips, points_x, points_y, point_score, image_width - 2 * half_patch_size, image_height, strip_count, half_patch_size);
    }
}
