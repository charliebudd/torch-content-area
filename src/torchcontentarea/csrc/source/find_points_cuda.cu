#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../common.hpp"

namespace cuda
{
    // =========================================================================
    // General functionality...
    __device__ float load_grayscale(const uint8* data, const int index, const int color_stride)
    {
        return 0.2126f * data[index + 0 * color_stride] + 0.7152f * data[index + 1 * color_stride] + 0.0722f * data[index + 2 * color_stride];
    }

    __device__ float sobel_filter(const float* data, const int index, const int x_stride, const int y_stride, float* x_grad, float* y_grad)
    {
        float left  = 0.25f * data[index - x_stride - y_stride] + 0.5f * data[index - x_stride] + 0.25f * data[index - x_stride + y_stride];
        float right = 0.25f * data[index + x_stride - y_stride] + 0.5f * data[index + x_stride] + 0.25f * data[index + x_stride + y_stride];
        *x_grad = right - left;

        float top = 0.25f * data[index - x_stride - y_stride] + 0.5f * data[index - y_stride] + 0.25f * data[index + x_stride - y_stride];
        float bot = 0.25f * data[index - x_stride + y_stride] + 0.5f * data[index + y_stride] + 0.25f * data[index + x_stride + y_stride];
        *y_grad = bot - top;

        return sqrt(*x_grad * *x_grad + *y_grad * *y_grad);
    }

    // =========================================================================
    // Kernels...

    __global__ void find_points_kernel(const uint8* g_image, int* g_edge_x, int* g_edge_y, float* g_edge_scores, const int image_width, const int image_height, const int strip_count, const FeatureThresholds feature_thresholds)
    {
        constexpr int warp_size = 32;

        int thread_count = blockDim.x;
        int warp_count = 1 + (thread_count - 1) / warp_size;

        extern __shared__ int s_shared_buffer[];
        float* s_image_strip = (float*)s_shared_buffer;
        int* s_cross_warp_operation_buffer = s_shared_buffer + 3 * thread_count;
        float* s_cross_warp_operation_buffer_2 = (float*)(s_shared_buffer + 3 * thread_count + warp_count);

        int warp_index = threadIdx.x >> 5;
        int lane_index = threadIdx.x & 31;

        bool flip = blockIdx.x == 1;

        // ============================================================
        // Load strip into shared memory...

        int image_x = flip ? image_width - 1 - threadIdx.x : threadIdx.x;

        int strip_index = blockIdx.y;
        int strip_height = 1 + (image_height - 2) / (1.0f + exp(-(strip_index - strip_count / 2.0f + 0.5f)/(strip_count / 8.0f)));
        
        #pragma unroll
        for (int y = 0; y < 3; y++)
        {
            int image_element_index = image_x + (strip_height + (y - 1)) * image_width;
            s_image_strip[threadIdx.x + y * thread_count] = load_grayscale(g_image, image_element_index, image_width * image_height);
        }
        
        __syncthreads();
        
        // ============================================================
        // Calculate largest preceeding intensity...

        float max_preceeding_intensity = s_image_strip[threadIdx.x + thread_count];

        #pragma unroll
        for (int d=1; d < 32; d<<=1) 
        {
            float other_intensity = __shfl_up_sync(0xffffffff, max_preceeding_intensity, d);

            if (lane_index >= d && other_intensity > max_preceeding_intensity) 
            {
                max_preceeding_intensity = other_intensity;
            }
        }

        if (lane_index == warp_size - 1)
        {
            s_cross_warp_operation_buffer[warp_index] = max_preceeding_intensity;
        }
        
        __syncthreads();

        if (warp_index == 0)
        {
            float warp_max = lane_index < warp_count ? s_cross_warp_operation_buffer[lane_index] : 0;

            #pragma unroll
            for (int d=1; d < 32; d<<=1) 
            {
                float other_max = __shfl_up_sync(0xffffffff, warp_max, d);

                if (lane_index >= d && other_max > warp_max) 
                {
                    warp_max = other_max;
                }
            }

            if (lane_index < warp_count)
            {
                s_cross_warp_operation_buffer[lane_index] = warp_max;
            }
        }

        __syncthreads();

        if (warp_index > 0)
        {
            float other_intensity = s_cross_warp_operation_buffer[warp_index-1];
            max_preceeding_intensity = other_intensity > max_preceeding_intensity ? other_intensity : max_preceeding_intensity;
        }

        // ============================================================
        // Applying sobel kernel to image patch...

        float x_grad = 0;
        float y_grad = 0;
        float grad = 0;

        if (threadIdx.x > 0 && threadIdx.x < thread_count - 1)
        {
            grad = sobel_filter(s_image_strip, threadIdx.x + thread_count, 1, thread_count, &x_grad, &y_grad);
        }
        
        // ============================================================
        // Calculating angle between gradient vector and center vector...

        float center_dir_x = (0.5f * image_width) - (float)image_x;
        float center_dir_y = (0.5f * image_height) - (float)strip_height;
        float center_dir_norm = sqrt(center_dir_x * center_dir_x + center_dir_y * center_dir_y);
    
        x_grad = flip ? -x_grad : x_grad;

        float dot = grad == 0 ? -1 : (center_dir_x * x_grad + center_dir_y * y_grad) / (center_dir_norm * grad);
        float angle = RAD2DEG * acos(dot);

        // ============================================================
        // Final scoring...

        float edge_score = tanh(grad / feature_thresholds.edge);
        float angle_score = 1.0f - tanh(angle / feature_thresholds.angle);
        float intensity_score = 1.0f - tanh(max_preceeding_intensity / feature_thresholds.intensity);

        float point_score = edge_score * angle_score * intensity_score;

        // ============================================================
        // Reduction to find the best edge...

        int best_edge_x = image_x;
        float best_edge_score = point_score;
        
        // warp reduction....
        #pragma unroll
        for (int offset = warp_size >> 1; offset > 0; offset >>= 1)
        {
            int other_edge_x = __shfl_down_sync(0xffffffff, best_edge_x, offset);
            float other_edge_score = __shfl_down_sync(0xffffffff, best_edge_score, offset);

            if (other_edge_score > best_edge_score)
            {
                best_edge_x = other_edge_x;
                best_edge_score = other_edge_score;
            }
        }

        if (lane_index == 0)
        {
            s_cross_warp_operation_buffer[warp_index] = best_edge_x;
            s_cross_warp_operation_buffer_2[warp_index] = best_edge_score;
        }

        __syncthreads();

        // block reduction....
        if (warp_index == 0 && lane_index < warp_count)
        {
            best_edge_x = s_cross_warp_operation_buffer[lane_index];
            best_edge_score = s_cross_warp_operation_buffer_2[lane_index];

            #pragma unroll
            for (int offset = warp_count >> 1 ; offset > 0; offset >>= 1)
            {
                int other_edge_x = __shfl_down_sync(0xffffffff, best_edge_x, offset);
                float other_edge_score = __shfl_down_sync(0xffffffff, best_edge_score, offset);

                if (other_edge_score > best_edge_score)
                {
                    best_edge_x = other_edge_x;
                    best_edge_score = other_edge_score;
                }
            }

            if (lane_index == 0)
            {
                int point_index = flip ? strip_index : strip_index + strip_count;
                
                if (best_edge_x < DISCARD_BORDER || best_edge_x >= image_width - DISCARD_BORDER)
                {
                    best_edge_score = 0.0f;
                }

                g_edge_x[point_index] = best_edge_x;
                g_edge_y[point_index] = strip_height;
                g_edge_scores[point_index] = best_edge_score;
            }
        }
    }

    // =========================================================================
    // Main function...
    
    void find_points(const uint8* image, const int image_height, const int image_width, const int strip_count, const FeatureThresholds feature_thresholds, int* points_x, int* points_y, float* point_scores)
    {
        int half_width = image_width / 2;
        int warps = 1 + (half_width - 1) / 32;
        int threads = warps * 32;

        dim3 grid(2, strip_count);
        dim3 block(threads);
        int  shared_memmory = (3 * threads + 2 * warps) * sizeof(int);

        find_points_kernel<<<grid, block, shared_memmory>>>(image, points_x, points_y, point_scores, image_width, image_height, strip_count, feature_thresholds);
    }
}
