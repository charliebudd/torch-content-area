#include <hip/hip_runtime.h>
#include "content_area_inference.cuh"

ContentAreaInference::ContentAreaInference()
{
    m_height_samples = 8;
    m_point_count = 2 * m_height_samples;
    m_buffer_size = (4 + 2 * 5 + 4 * m_point_count) * sizeof(uint);

    hipHostMalloc(&m_hst_buffer, m_buffer_size);
    m_hst_edge_x = (uint*)m_hst_buffer + 0 * m_point_count;
    m_hst_edge_y = (uint*)m_hst_buffer + 1 * m_point_count;
    m_hst_norm_x = (float*)m_hst_buffer + 2 * m_point_count;
    m_hst_norm_y = (float*)m_hst_buffer + 3 * m_point_count;
    m_hst_x_sums = (float*)m_hst_buffer + 4 * m_point_count;
    m_hst_xx_sums = (float*)m_hst_buffer + (4 * m_point_count + 5);
    m_hst_circle = (float*)m_hst_buffer + (4 * m_point_count + 2 * 5);

    hipMalloc(&m_dev_buffer, m_buffer_size);
    m_dev_edge_x = (uint*)m_dev_buffer + 0 * m_point_count;
    m_dev_edge_y = (uint*)m_dev_buffer + 1 * m_point_count;
    m_dev_norm_x = (float*)m_dev_buffer + 2 * m_point_count;
    m_dev_norm_y = (float*)m_dev_buffer + 3 * m_point_count;
    m_dev_x_sums = (float*)m_dev_buffer + 4 * m_point_count;
    m_dev_xx_sums = (float*)m_dev_buffer + (4 * m_point_count + 5);
    m_dev_circle = (float*)m_dev_buffer + (4 * m_point_count + 2 * 5);
}

ContentAreaInference::~ContentAreaInference()
{
    hipFree(m_dev_buffer);
    hipHostFree(m_hst_buffer);
}

// ContentArea ContentAreaInference::infer_area(uint8* image, const uint image_height, const uint image_width)
// {
//     ### Defined in "infer_area.cu" ###
// }

// void ContentAreaInference::draw_area(const ContentArea area, uint8* mask, const uint mask_height, const uint mask_width)
// {
//     ### Defined in "draw_area.cu" ###
// }

// void ContentAreaInference::crop_area(const ContentArea area, const uint8* src_image, uint8* dst_image, const uint src_width, const uint src_height, const uint dst_width, const uint dst_height, const InterpolationMode interpolation_mode)
// {
//     ### Defined in "crop_area.cu" ###
// }
