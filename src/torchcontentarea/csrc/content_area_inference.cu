#include <hip/hip_runtime.h>
#include "content_area_inference.cuh"

ContentAreaInference::ContentAreaInference()
{
    m_height_samples = 16;
    m_point_count = 2 * m_height_samples;
    m_buffer_size = 10 * (3 * m_point_count + 4) * sizeof(uint);

    hipHostMalloc(&m_hst_buffer, m_buffer_size);
    m_hst_edge_x =      (uint*) m_hst_buffer + 0 * m_point_count;
    m_hst_edge_y =      (uint*) m_hst_buffer + 1 * m_point_count;
    m_hst_edge_scores = (float*)m_hst_buffer + 2 * m_point_count;
    m_hst_circle =      (float*)m_hst_buffer + 3 * m_point_count;

    hipMalloc(&m_dev_buffer, m_buffer_size);
    m_dev_edge_x =      (uint*) m_dev_buffer + 0 * m_point_count;
    m_dev_edge_y =      (uint*) m_dev_buffer + 1 * m_point_count;
    m_dev_edge_scores = (float*)m_dev_buffer + 2 * m_point_count;
    m_dev_circle =      (float*)m_dev_buffer + 3 * m_point_count;
}

ContentAreaInference::~ContentAreaInference()
{
    hipFree(m_dev_buffer);
    hipHostFree(m_hst_buffer);
}

// ContentArea ContentAreaInference::infer_area(uint8* image, const uint image_height, const uint image_width)
// {
//     ### Defined in "infer_area.cu" ###
// }

// void ContentAreaInference::draw_area(const ContentArea area, uint8* mask, const uint mask_height, const uint mask_width)
// {
//     ### Defined in "draw_area.cu" ###
// }

// void ContentAreaInference::crop_area(const ContentArea area, const uint8* src_image, uint8* dst_image, const uint src_width, const uint src_height, const uint dst_width, const uint dst_height, const InterpolationMode interpolation_mode)
// {
//     ### Defined in "crop_area.cu" ###
// }
