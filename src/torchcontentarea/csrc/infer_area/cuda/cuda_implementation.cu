#include <torch/extension.h>
#include "infer_area_cuda.cuh"

// #include "../cpu/infer_area_cpu.h"

torch::Tensor InferAreaHandcrafted::cuda_implementation(torch::Tensor image, uint strip_count, FeatureThresholds feature_thresholds, ConfidenceThresholds confidence_thresholds)
{
    check_image_tensor(image);

    uint batch_count = image.size(0);
    uint image_channels = image.size(1);
    uint image_height = image.size(2);
    uint image_width = image.size(3);
    uint point_count = 2 * strip_count;

    torch::Tensor result = torch::empty({batch_count, 4}, torch::device(image.device()).dtype(torch::kFloat32));

    void* temp_buffer;
    hipMalloc(&temp_buffer, 3 * batch_count * point_count * sizeof(uint));
    uint*  points_x = (uint*) temp_buffer + 0 * batch_count * point_count; 
    uint*  points_y = (uint*) temp_buffer + 1 * batch_count * point_count; 
    float* points_s = (float*)temp_buffer + 2 * batch_count * point_count;

    // void* cpu_temp_buffer = malloc(3 * batch_count * point_count * sizeof(uint));
    // uint*  cpu_points_x = (uint*) cpu_temp_buffer + 0 * batch_count * point_count;
    // uint*  cpu_points_y = (uint*) cpu_temp_buffer + 1 * batch_count * point_count;
    // float* cpu_points_s = (float*)cpu_temp_buffer + 2 * batch_count * point_count;
    // image = image.cpu();
    // find_points_cpu(image.data_ptr<uint8>(), image_height, image_width, strip_count, feature_thresholds, cpu_points_x, cpu_points_y, cpu_points_s);
    // image = image.cuda();
    // hipMemcpy(temp_buffer, cpu_temp_buffer, 3 * batch_count * point_count * sizeof(uint), hipMemcpyHostToDevice);
    // delete cpu_temp_buffer;

    find_points(image.data_ptr<uint8>(), image_height, image_width, strip_count, feature_thresholds, points_x, points_y, points_s);
    fit_circle(points_x, points_y, points_s, point_count, confidence_thresholds, image_height, image_width, result.data_ptr<float>());

    hipFree(temp_buffer);

    return result;
}

torch::Tensor InferAreaLearned::cuda_implementation(torch::Tensor image, uint strip_count, torch::jit::Module model, uint model_patch_size, ConfidenceThresholds confidence_thresholds)
{
    check_image_tensor(image);

    uint batch_count = image.size(0);
    uint image_channels = image.size(1);
    uint image_height = image.size(2);
    uint image_width = image.size(3);
    uint point_count = 2 * strip_count;

    torch::Tensor result = torch::empty({batch_count, 4}, torch::device(image.device()).dtype(torch::kFloat32));
    torch::Tensor strips = torch::empty({batch_count * strip_count, 5, model_patch_size, image_width}, torch::device(image.device()).dtype(torch::kFloat32));
    std::vector<torch::jit::IValue> model_input = {strips};

    void* temp_buffer;
    hipMalloc(&temp_buffer, 3 * batch_count * point_count * sizeof(uint));
    uint*  points_x = (uint*) temp_buffer + 0 * batch_count * point_count; 
    uint*  points_y = (uint*)temp_buffer + 1 * batch_count * point_count;
    float* points_s = (float*)temp_buffer + 2 * batch_count * point_count; 
    
    make_strips(image.data_ptr<uint8>(), image_height, image_width, strip_count, model_patch_size, strips.data_ptr<float>());
    
    torch::Tensor strip_scores = torch::sigmoid(model.forward(model_input).toTensor());

    find_points_from_strip_scores(strip_scores.data_ptr<float>(), image_height, image_width, strip_count, model_patch_size, points_x, points_y, points_s);
    
    fit_circle(points_x, points_y, points_s, point_count, confidence_thresholds, image_height, image_width, result.data_ptr<float>());

    hipFree(temp_buffer);

    return result;
}
